#include "hip/hip_runtime.h"
// Patrik Tennberg, 2012

#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
  return;
}

typedef unsigned long long int uint64;

__global__ void setup_kernel(hiprandState *globalStates) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  hiprand_init(tid, tid, 0, &globalStates[tid]);
}

template <int num_threads, int points_per_thread>
__global__ void inside_circle_kernel(hiprandState *globalStates,
                                     uint64 *per_block_results) {
  __shared__ uint64 output[num_threads];

  const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  hiprandState localState = globalStates[tid];

  uint64 inside = 0;

  for (int i = 0; i < points_per_thread; i++) {
    const float x = hiprand_uniform(&localState);
    const float y = hiprand_uniform(&localState);

    if (x * x + y * y <= 1.0) {
      inside++;
    }
  }
  globalStates[tid] = localState;
  output[threadIdx.x] = inside;
  // wait until all threads in the block have updated their partial sums
  __syncthreads();

  // thread 0 of every block writes the final result
  if (threadIdx.x == 0) {
    inside = 0;
    for (int i = 0; i < blockDim.x; i++) {
      inside += output[i];
    }
    per_block_results[blockIdx.x] = inside;
  }
}

__global__ void reduction_kernel(uint64 *result,
                                 const uint64 *per_block_results, size_t N) {
  uint64 partialSum = 0;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    partialSum += per_block_results[i];
  }
  atomicAdd(result, partialSum);
}

uint64 verify_result(const uint64 gpu_result, const uint64 *d_per_block_results,
                     const size_t N) {
  uint64 h_per_block_results[N];

  CudaSafeCall(hipMemcpy(h_per_block_results, d_per_block_results,
                          sizeof(h_per_block_results), hipMemcpyDeviceToHost));

  uint64 total_inside = 0;

  for (int i = 0; i < N; i++) {
    total_inside += h_per_block_results[i];
  }
  if (gpu_result != total_inside) {
    printf("Diff: %lld != %lld\n", gpu_result, total_inside);
  }
  return total_inside;
}

int main(void) {
  const int N = 600000;  // Total number of threads
  const int points_per_thread = 2048;
  const unsigned int num_threads = 1024;
  const unsigned int num_blocks = (N + num_threads - 1) / num_threads;
  const unsigned int red_num_blocks =
      (num_blocks + num_threads - 1) / num_threads;
  const uint64 scenarious = points_per_thread * num_threads * num_blocks;

  // Initialize hiprand
  hiprandState *d_devStates;

  CudaSafeCall(
      hipMalloc(&d_devStates, num_blocks * num_threads * sizeof(hiprandState)));

  setup_kernel<<<num_blocks, num_threads>>>(d_devStates);
  CudaSafeCall(
      hipDeviceSynchronize());  // We need to wait since the kernel can timeout
                                 // Estimate Pi
  uint64 *d_per_block_results;

  CudaSafeCall(hipMalloc(&d_per_block_results,
                          num_blocks * sizeof(*d_per_block_results)));
  CudaSafeCall(hipMemset(d_per_block_results, 0,
                          num_blocks * sizeof(*d_per_block_results)));

  printf("Estimating PI with %lld scenarious using %u blocks and %u threads\n",
         scenarious, num_blocks, num_threads);
  inside_circle_kernel<num_threads,
                       points_per_thread><<<num_blocks, num_threads>>>(
      d_devStates, d_per_block_results);
  CudaSafeCall(hipDeviceSynchronize());
  // Reduce partial sums
  uint64 *d_total_inside;

  CudaSafeCall(hipMalloc(&d_total_inside, sizeof(uint64)));
  CudaSafeCall(hipMemset(d_total_inside, 0, sizeof(uint64)));

  printf("Reducing %u values, using %u block(s) and %u threads\n", num_blocks,
         red_num_blocks, num_threads);
  reduction_kernel<<<red_num_blocks, num_threads>>>(
      d_total_inside, d_per_block_results, num_blocks);
  CudaSafeCall(hipDeviceSynchronize());
  // Verify result from kernels
  uint64 total_inside = 0;

  CudaSafeCall(hipMemcpy(&total_inside, d_total_inside, sizeof(total_inside),
                          hipMemcpyDeviceToHost));

  verify_result(total_inside, d_per_block_results, num_blocks);
  // Display our Pi estimate
  const double pi = 4.0 * double(total_inside) / scenarious;

  printf("estimated pi = %f, expected = 3.141592653589793238462643\n", pi);

  CudaSafeCall(hipFree(d_devStates));
  CudaSafeCall(hipFree(d_per_block_results));
  CudaSafeCall(hipFree(d_total_inside));

  return 0;
}